
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(1);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Sequential Radix Sort
void sortByHost(const uint32_t * in, int n,
                uint32_t * out)
{
    int * bits = (int *)malloc(n * sizeof(int));
    int * nOnesBefore = (int *)malloc(n * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // To free memory later
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    // Loop from LSB (Least Significant Bit) to MSB (Most Significant Bit)
	// In each loop, sort elements according to the current bit from src to dst 
	// (using STABLE counting sort)
    for (int bitIdx = 0; bitIdx < sizeof(uint32_t) * 8; bitIdx++)
    {
        // Extract bits
        for (int i = 0; i < n; i++)
            bits[i] = (src[i] >> bitIdx) & 1;

        // Compute nOnesBefore
        nOnesBefore[0] = 0;
        for (int i = 1; i < n; i++)
            nOnesBefore[i] = nOnesBefore[i-1] + bits[i-1];

        // Compute rank and write to dst
        int nZeros = n - nOnesBefore[n-1] - bits[n-1];
        for (int i = 0; i < n; i++)
        {
            int rank;
            if (bits[i] == 0)
                rank = i - nOnesBefore[i];
            else
                rank = nZeros + nOnesBefore[i];
            dst[rank] = src[i];
        }

        // Swap src and dst
        uint32_t * temp = src;
        src = dst;
        dst = temp;
    }

    // Does out array contain results?
    memcpy(out, src, n * sizeof(uint32_t));

    // Free memory
    free(originalSrc);
    free(bits);
    free(nOnesBefore);
}

// Kernel 1: Extract bit at position `bitIdx`
__global__ void extract_bits_kernel(uint32_t *a, int *bit, int n, int bitIdx) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        bit[tid] = (a[tid] >> bitIdx) & 1;
    }
}

// Kernel 3: Sort elements based on the current bit
__global__ void sort_by_bit_kernel(uint32_t *a, uint32_t *out, int *bit, int *nOneBefore, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < n) {
        int numZeros = n - nOneBefore[n - 1] - bit[n - 1];
        int rank;
        if (bit[tid] == 0) {
            rank = tid - nOneBefore[tid];
        } else {
            rank = numZeros + nOneBefore[tid];
        }
        out[rank] = a[tid];
    }
}


__global__ void scanBlkKernel2(int * in, int n, int * out, int * blkSums)
{
	// 1. Each block loads data from GMEM to SMEM
	extern __shared__ int s_data[];
	int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	int i2 = i1 + blockDim.x;
	if (i1 < n)
		s_data[threadIdx.x] = in[i1];
	if (i2 < n)
		s_data[threadIdx.x + blockDim.x] = in[i2];
	__syncthreads();

	// 2. Each block does scan with data on SMEM
	// 2.1. Reduction phase
	for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
	{
		int s_dataIdx = (threadIdx.x + 1) * 2 * stride - 1; // To avoid warp divergence
		if (s_dataIdx < 2 * blockDim.x)
			s_data[s_dataIdx] += s_data[s_dataIdx - stride];
		__syncthreads();
	}
	// 2.2. Post-reduction phase
	for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		int s_dataIdx = (threadIdx.x + 1) * 2 * stride - 1 + stride; // Wow
		if (s_dataIdx < 2 * blockDim.x)
			s_data[s_dataIdx] += s_data[s_dataIdx - stride];
		__syncthreads();
	}

	// 3. Each block writes results from SMEM to GMEM
	if (i1 < n)
		out[i1] = s_data[threadIdx.x];
	if (i2 < n)
		out[i2] = s_data[threadIdx.x + blockDim.x];

	if (blkSums != NULL && threadIdx.x == 0)
		blkSums[blockIdx.x] = s_data[2 * blockDim.x - 1];
}

__global__ void addPrevBlkSum(int * blkSumsScan, int * blkScans, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
    if (i < n)
        blkScans[i] += blkSumsScan[blockIdx.x];
}

void scan(int * d_in1, int n, int * d_out2, dim3 blkSize=dim3(1)) {
    int blkDataSize;
    blkDataSize = 2 * blkSize.x;
    // 1. Scan locally within each block, 
    //    and collect blocks' sums into array
    int * d_blkSums;
    dim3 gridSize((n - 1) / blkDataSize + 1);
    if (gridSize.x > 1)
    {
        CHECK(hipMalloc(&d_blkSums, gridSize.x * sizeof(int)));
    }
    else
    {
        d_blkSums = NULL;
    }

    size_t smem = blkDataSize * sizeof(int);
    scanBlkKernel2<<<gridSize, blkSize, smem>>>(d_in1, n, d_out2, d_blkSums);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    if (gridSize.x > 1)
    {
        // 2. Compute each block's previous sum 
        //    by scanning array of blocks' sums
        size_t temp = gridSize.x * sizeof(int);
        int * blkSums = (int*)malloc(temp);
        CHECK(hipMemcpy(blkSums, d_blkSums, temp, hipMemcpyDeviceToHost));
        for (int i = 1; i < gridSize.x; i++)
            blkSums[i] += blkSums[i-1];
        CHECK(hipMemcpy(d_blkSums, blkSums, temp, hipMemcpyHostToDevice));

        // 3. Add each block's previous sum to its scan result in step 1
        addPrevBlkSum<<<gridSize.x - 1, blkDataSize>>>(d_blkSums, d_out2, n);
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        
        free(blkSums);
    }

    CHECK(hipFree(d_blkSums));
}

void sortByDevice(const uint32_t *in, int n, uint32_t *out, int blockSize) {
    uint32_t *d_in, *d_out;
    int *d_bit, *d_nOneBefore;

    // Allocate device memory
    hipMalloc(&d_in, n * sizeof(uint32_t));
    hipMalloc(&d_out, n * sizeof(uint32_t));
    hipMalloc(&d_bit, n * sizeof(int));
    hipMalloc(&d_nOneBefore, (n + 1) * sizeof(int));

    hipMemcpy(d_in, in, n * sizeof(uint32_t), hipMemcpyHostToDevice);

    int numBlocks = (n + blockSize - 1) / blockSize;

    for (int bitIdx = 0; bitIdx < 32; ++bitIdx) {
        // Step 1: Extract bits (kernel)
        extract_bits_kernel<<<numBlocks, blockSize>>>(d_in, d_bit, n, bitIdx);
        hipDeviceSynchronize();

        // Step 2: Perform exclusive scan sequentially on host
        hipMemset(d_nOneBefore, 0, sizeof(int));

        dim3 blkSize = dim3(blockSize);
        scan(d_bit, n, d_nOneBefore + 1, blkSize);

        // Step 3: Sort elements based on the current bit (kernel)
        sort_by_bit_kernel<<<numBlocks, blockSize>>>(d_in, d_out, d_bit, d_nOneBefore, n);
        hipDeviceSynchronize();

        // Swap input and output arrays for the next iteration
        uint32_t *temp = d_in;
        d_in = d_out;
        d_out = temp;
    }

    // Copy sorted result back to host
    hipMemcpy(out, d_in, n * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_bit);
    hipFree(d_nOneBefore);
}

// Radix Sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        bool useDevice=false, int blockSize=1)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix Sort by host\n");
        sortByHost(in, n, out);
    }
    else // use device
    {
    	printf("\nRadix Sort by device\n");
        sortByDevice(in, n, out, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    // int n = 50; // For test by eye
    int n = (1 << 24) + 1;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
    {
        in[i] = rand() % 255; // For test by eye
        // in[i] = rand();
    }
    //printArray(in, n); // For test by eye

    // DETERMINE BLOCK SIZE
    int blockSize = 512; // Default 
    if (argc == 2)
        blockSize = atoi(argv[1]);

    // SORT BY HOST
    sort(in, n, correctOut);
    // printArray(correctOut, n); // For test by eye
    
    // SORT BY DEVICE
    sort(in, n, out, true, blockSize);
    // printArray(out, n); // For test by eye
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
