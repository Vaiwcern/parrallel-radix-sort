
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream> 

using namespace std;

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(1);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start); 
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

/*
Scan within each block's data (work-efficient), write results to "out", and
write each block's sum to "blkSums" if "blkSums" is not NULL.
*/
__global__ void scanBlkKernel2(int * in, int n, int * out, int * blkSums)
{
    // TODO
	// 1. Each block loads data from GMEM to SMEM
	extern __shared__ int s_data[];
	int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	int i2 = i1 + blockDim.x;
	if (i1 < n)
		s_data[threadIdx.x] = in[i1];
	if (i2 < n)
		s_data[threadIdx.x + blockDim.x] = in[i2];
	__syncthreads();

	// 2. Each block does scan with data on SMEM
	// 2.1. Reduction phase
	for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
	{
		int s_dataIdx = (threadIdx.x + 1) * 2 * stride - 1; // To avoid warp divergence
		if (s_dataIdx < 2 * blockDim.x)
			s_data[s_dataIdx] += s_data[s_dataIdx - stride];
		__syncthreads();
	}
	// 2.2. Post-reduction phase
	for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		int s_dataIdx = (threadIdx.x + 1) * 2 * stride - 1 + stride; // Wow
		if (s_dataIdx < 2 * blockDim.x)
			s_data[s_dataIdx] += s_data[s_dataIdx - stride];
		__syncthreads();
	}

	// 3. Each block writes results from SMEM to GMEM
	if (i1 < n)
		out[i1] = s_data[threadIdx.x];
	if (i2 < n)
		out[i2] = s_data[threadIdx.x + blockDim.x];

	if (blkSums != NULL && threadIdx.x == 0)
		blkSums[blockIdx.x] = s_data[2 * blockDim.x - 1];
}

// TODO: You can define necessary functions here
__global__ void addPrevBlkSum(int * blkSumsScan, int * blkScans, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
    if (i < n)
        blkScans[i] += blkSumsScan[blockIdx.x];
}

void scan(int * in1, int n, int * out1, dim3 blkSize=dim3(1)) {
    int blkDataSize;
    printf("\nScan by device, work-efficient\n");
    blkDataSize = 2 * blkSize.x;
    // 1. Scan locally within each block, 
    //    and collect blocks' sums into array
    
    int * d_in1, * d_out2, * d_blkSums;
    size_t nBytes = n * sizeof(int);
    CHECK(hipMalloc(&d_in1, nBytes)); 
    CHECK(hipMalloc(&d_out2, nBytes)); 
    dim3 gridSize((n - 1) / blkDataSize + 1);
    if (gridSize.x > 1)
    {
        CHECK(hipMalloc(&d_blkSums, gridSize.x * sizeof(int)));
    }
    else
    {
        d_blkSums = NULL;
    }

    CHECK(hipMemcpy(d_in1, in1, nBytes, hipMemcpyHostToDevice));

    size_t smem = blkDataSize * sizeof(int);
    scanBlkKernel2<<<gridSize, blkSize, smem>>>(d_in1, n, d_out2, d_blkSums);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    if (gridSize.x > 1)
    {
        // 2. Compute each block's previous sum 
        //    by scanning array of blocks' sums
        // TODO
        size_t temp = gridSize.x * sizeof(int);
        int * blkSums = (int*)malloc(temp);
        CHECK(hipMemcpy(blkSums, d_blkSums, temp, hipMemcpyDeviceToHost));
        for (int i = 1; i < gridSize.x; i++)
            blkSums[i] += blkSums[i-1];
        CHECK(hipMemcpy(d_blkSums, blkSums, temp, hipMemcpyHostToDevice));

        // 3. Add each block's previous sum to its scan result in step 1
        addPrevBlkSum<<<gridSize.x - 1, blkDataSize>>>(d_blkSums, d_out2, n);
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        
        free(blkSums);
    }

    CHECK(hipMemcpy(out1, d_out2, nBytes, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_in1));
    CHECK(hipFree(d_out2));
    CHECK(hipFree(d_blkSums));
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(int * out, int * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = 5;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(int);
    int * in = (int *)malloc(bytes);
    int * out = (int *)malloc(bytes); // Device result
    int * correctOut = (int *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = (int)(rand() & 0xFF); // random int in [-127, 128]

    // DETERMINE BLOCK SIZE
    dim3 blockSize1(512); 
    dim3 blockSize2(256); 
    if (argc == 3)
    {
        blockSize1.x = atoi(argv[1]);
        blockSize2.x = atoi(argv[2]);
    }

    for (int i = 0; i < n; ++i) {
        cout << in[i] << "\n";
    }

    // SCAN BY DEVICE, WORK-EFFICIENT
    memset(out, 0, n * sizeof(int)); // Reset out
    scan(in, n, out, blockSize2);

    for (int i = 0; i < n; ++i) {
        cout << out[i] << "\n";
    }

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
